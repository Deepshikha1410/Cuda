#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void find_primes(int* numbers, int num_elements, int* prime_count) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < num_elements; i += stride) {
    if (numbers[i] <= 1) continue;

    int is_prime = 1;
    for (int j = 2; j * j <= numbers[i]; j++) {
      if (numbers[i] % j == 0) {
        is_prime = 0;
        break;
      }
    }

    if (is_prime) {
      atomicAdd(prime_count, 1);
    }
  }
}

int main() {
  const int num_elements = 100001;

  // Allocate memory on host for numbers and prime count
  int* numbers = (int*)malloc(num_elements * sizeof(int));
  for (int i = 0; i < num_elements; i++) {
    numbers[i] = i;
  }
  int* prime_count = (int*)malloc(sizeof(int));
  *prime_count = 0;

  // Allocate memory on device for numbers and prime count
  int* d_numbers;
  int* d_prime_count;
  hipMalloc(&d_numbers, num_elements * sizeof(int));
  hipMalloc(&d_prime_count, sizeof(int));

  // Copy data from host to device
  hipMemcpy(d_numbers, numbers, num_elements * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_prime_count, prime_count, sizeof(int), hipMemcpyHostToDevice);

  // Define grid and block size for kernel execution
  int threadsPerBlock = 256;
  int numBlocks = (num_elements + threadsPerBlock - 1) / threadsPerBlock;

  // Declare elapsed_time
  float elapsed_time;

  // Start timer
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Launch kernel
  find_primes<<<numBlocks, threadsPerBlock>>>(d_numbers, num_elements, d_prime_count);

  // Wait for GPU to finish
  hipDeviceSynchronize();

  // Stop timer
  hipEventRecord(stop, 0);
  hipEventElapsedTime(&elapsed_time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Copy prime count back from device to host
  hipMemcpy(prime_count, d_prime_count, sizeof(int), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_numbers);
  hipFree(d_prime_count);

  // Print results
  printf("Found %d prime numbers between 1 and 100001.\n", *prime_count);
  printf("Execution time on GPU: %.3f ms\n", elapsed_time);

  // Free host memory
  free(numbers);
  free(prime_count);

  return 0;
}