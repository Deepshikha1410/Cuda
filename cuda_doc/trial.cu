#include <stdio.h>
#include <hip/hip_runtime.h>


#define M 2  // Array rows
#define N 3 // Array columns

__global__ void addArrays2D(int* d_result, int* d_array1, int* d_array2) {
  int blockIdxX = blockIdx.x;
  int blockIdxY = blockIdx.y;
  int threadIdxX = threadIdx.x;
  int threadIdxY = threadIdx.y;

  // Calculate a unique thread ID within a block
  int uniqueThreadId = blockIdxX * blockDim.x * blockDim.y + 
                       blockIdxY * blockDim.x + 
                       threadIdxX + threadIdxY;

  int row = blockIdxY * blockDim.y + threadIdxY;
  int col = blockIdxX * blockDim.x + threadIdxX;

  // Check for valid element within array bounds
  if (row < M && col < N) {
    int result = d_array1[row * N + col] + d_array2[row * N + col];
    d_result[row * N + col] = result;
    // Print the thread ID along with operation details 
    printf("Glocally thread usnique id is : Thread %d: Adding A[%d][%d] + B[%d][%d] = Result[%d][%d]\n", 
           uniqueThreadId, row, col, row, col, row, col);
  }
}

int main() {
  // Host memory for the arrays
  int host_array1[M][N] = {
    {1, 2, 3},
    {6, 7, 8},
 
  };
  int host_array2[M][N] = {
    {10, 20, 30},
    {60, 70, 80},

  };

  // Allocate memory on device for the arrays
  int* d_array1, *d_array2, *d_result;
  hipMalloc(&d_array1, M * N * sizeof(int));
  hipMalloc(&d_array2, M * N * sizeof(int));
  hipMalloc(&d_result, M * N * sizeof(int));

  // Copy arrays from host to device
  hipMemcpy(d_array1, host_array1, M * N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_array2, host_array2, M * N * sizeof(int), hipMemcpyHostToDevice);

  // Define grid and block sizes
  int threadsPerBlockX = 6;
  int threadsPerBlockY = 3;
  dim3 threadsPerBlock(threadsPerBlockX, threadsPerBlockY, 1);
  int numBlocksX = (N + threadsPerBlockX - 1) / threadsPerBlockX;
  int numBlocksY = (M + threadsPerBlockY - 1) / threadsPerBlockY;
  dim3 blocksPerGrid(numBlocksX, numBlocksY, 1);

  // Launch the kernel
  addArrays2D<<<blocksPerGrid, threadsPerBlock>>>(d_result, d_array1, d_array2);

  // Synchronize threads

  hipDeviceSynchronize();

  // Allocate memory on host to store results
  int host_result[M][N];

  // Copy results back from device to host
  hipMemcpy(host_result, d_result, M * N * sizeof(int), hipMemcpyDeviceToHost);

  // Print the result array
  printf("Resulting Array:\n");
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N;++j) {
      printf("%d ", host_result[i][j]);
    }
     printf("\n");
  }

   // Free memory on device
   hipFree(d_array1);
   hipFree(d_array2);
   hipFree(d_result);

  return 0;
}


