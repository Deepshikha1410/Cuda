#include<hip/hip_runtime.h>

#include<stdio.h>
#define arraySize 5
int main(){
    float hostArray[arraySize] = {1.0, 2.0, 3.0, 4.0, 5.0 };
    float resultArray[arraySize];

    float* deviceArray;
    hipMalloc((void **)&deviceArray, arraySize * sizeof(float));

    hipMemcpy(deviceArray, hostArray, arraySize * sizeof(float) ,hipMemcpyHostToDevice);

    hipMemcpy(resultArray, deviceArray, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    //Display results using printf
    printf("Original Array: ");
    for(int i =0; i < arraySize; ++i){
        printf("%f", hostArray[i]);
    }
    printf("\n");

    printf("Copied Array: ");
    for(int i = 0; i < arraySize; ++i){
        printf("%f", resultArray[i]);
    }
    printf("\n");

    //Free allocated memory on GPU
    hipFree(deviceArray);

    return 0;
}