#include "hip/hip_runtime.h"

#include<stdio.h>

//CUDA kernel to add two integers
__global__ void addIntegers(int* a, int* b, int* result){
    *result = *a + *b;
}

int main(){
    // Host variables
    int host_a = 5;
    int host_b = 7;
    int host_result =0;

//Device variables
int* device_a, * device_b, * device_result;

//Allocate memory on the devoice
hipMalloc((void**)&device_a, sizeof(int));
hipMalloc((void**)&device_b, sizeof(int));
hipMalloc((void**)&device_result, sizeof(int));

//copy data from host to device
hipMemcpy(device_a, &host_a, sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(device_b, &host_b, sizeof(int), hipMemcpyHostToDevice);

//Launch the kernel with one block and one thread
addIntegers <<<1, 1>>> (device_a, device_b, device_result);

//copy the result from device to host 
hipMemcpy(&host_result, device_result, sizeof(int), hipMemcpyDeviceToHost);

//display the result
printf("Sum of %d and %d is %d\n",host_a, host_b, host_result);

//free allocated memory
hipFree(device_a);
hipFree(device_b);
hipFree(device_result);

return 0;
}