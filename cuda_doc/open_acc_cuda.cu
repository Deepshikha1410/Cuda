#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 10000  // Adjust array size as needed

__global__ void elementWiseSum(float *A, float *B) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    A[i] = A[i] + B[i];
  }
}

__global__ void squareElements(float *A) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    A[i] = A[i] * A[i];
  }
}

int main() {
  // Allocate memory on the host for arrays A and B
  float *A_h, *B_h;
  hipMalloc(&A_h, N * sizeof(float));
  hipMalloc(&B_h, N * sizeof(float));

  // Initialize arrays A and B on the host (assuming initialization is not performance-critical)
  for (int i = 0; i < N; ++i) {
    A_h[i] = i * 0.1f;
    B_h[i] = i * 0.1f;
  }

  // Allocate memory on the device (GPU) for arrays A and B
  float *A_d, *B_d;
  hipMalloc(&A_d, N * sizeof(float));
  hipMalloc(&B_d, N * sizeof(float));

  // Start timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Transfer data from host to device
  hipMemcpy(A_d, A_h, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, N * sizeof(float), hipMemcpyHostToDevice);

  // Launch element-wise sum kernel
  int threadsPerBlock = 256;  // Adjust as needed based on GPU architecture
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  elementWiseSum<<<blocksPerGrid, threadsPerBlock>>>(A_d, B_d);
  hipDeviceSynchronize();  // Wait for kernel to finish

  // Launch square elements kernel
  squareElements<<<blocksPerGrid, threadsPerBlock>>>(A_d);
  hipDeviceSynchronize();  // Wait for kernel to finish

  // Transfer data back from device to host
  hipMemcpy(A_h, A_d, N * sizeof(float), hipMemcpyDeviceToHost);

  // Stop timing
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed_time_ms;
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  hipFree(start);
  hipFree(stop);

  // Print results
  printf("Required elements of (A+B):\n");
  // ... (print results as before)

  // Free memory on the host and device
  hipFree(A_h);
  hipFree(B_h);
  hipFree(A_d);
  hipFree(B_d);

  return 0;
}
